#include "hip/hip_runtime.h"
#include "stereotgv.h"

__global__ void TgvConvertDisparityToDepthKernel(float *disparity, float baseline,
	float focal, int width, int height, int stride, float *depth)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	const int pos = ix + iy * stride;

	if (ix >= width || iy >= height) return;

	/*float Z = baseline * focal / disparity[pos];
	float X = (ix - width / 2)*Z / focal;
	float Y = (iy - height / 2)*Z / focal;
	depth[pos] = sqrt(Z * Z + X * X + Y * Y);*/
	depth[pos] = baseline * focal / disparity[pos];
}


void StereoTgv::ConvertDisparityToDepth(float *disparity, float baseline, float focal, int w, int h, int s, float *depth)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	TgvConvertDisparityToDepthKernel << <blocks, threads >> > (disparity, baseline, focal, w, h, s, depth);
}


//*******************
// Masked
//*******************
__global__ void TgvConvertDisparityToDepthMaskedKernel(float *disparity, float* mask, float baseline,
	float focal, int width, int height, int stride, float *depth)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	if ((iy >= height) && (ix >= width)) return;
	int pos = ix + iy * stride;
	if (mask[pos] == 0.0f) return;

	depth[pos] = baseline * focal / disparity[pos];
}


void StereoTgv::ConvertDisparityToDepthMasked(float *disparity, float* mask, float baseline, float focal, 
	int w, int h, int s, float *depth)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	TgvConvertDisparityToDepthMaskedKernel << <blocks, threads >> > (disparity, mask, baseline, focal, w, h, s, depth);
}