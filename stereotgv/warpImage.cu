#include "hip/hip_runtime.h"
#include "stereotgv.h"

/// image to warp
texture<float, hipTextureType2D, hipReadModeElementType> texToWarp;
texture<float2, hipTextureType2D, hipReadModeElementType> texTv;
texture<float, hipTextureType2D, hipReadModeElementType> texTvx;
texture<float, hipTextureType2D, hipReadModeElementType> texTvy;

__global__ void TgvWarpingKernel(int width, int height, int stride,
	const float2 *warpUV, float *out)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	const int pos = ix + iy * stride;

	if (ix >= width || iy >= height) return;

	float x = ((float)ix + warpUV[pos].x + 0.5f) / (float)width;
	float y = ((float)iy + warpUV[pos].y + 0.5f) / (float)height;

	out[pos] = tex2D(texToWarp, x, y);
}

void StereoTgv::WarpImage(const float *src, int w, int h, int s,
	const float2 *warpUV, float *out)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	// mirror if a coordinate value is out-of-range
	texToWarp.addressMode[0] = hipAddressModeMirror;
	texToWarp.addressMode[1] = hipAddressModeMirror;
	texToWarp.filterMode = hipFilterModeLinear;
	texToWarp.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipBindTexture2D(0, texToWarp, src, w, h, s * sizeof(float));

	TgvWarpingKernel << <blocks, threads >> > (w, h, s, warpUV, out);
}

// **************************************************
// ** Find Warping vector direction (tvx2, tvy2) for Fisheye Stereo
// **************************************************

__global__ void TgvFindWarpingVectorKernel(const float2 *warpUV,
	int width, int height, int stride, float2 *tvx2)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	const int pos = ix + iy * stride;

	if (ix >= width || iy >= height) return;

	float x = ((float)ix + warpUV[pos].x + 0.5f) / (float)width;
	float y = ((float)iy + warpUV[pos].y + 0.5f) / (float)height;

	tvx2[pos].x = tex2D(texTvx, x, y);
	tvx2[pos].x = tex2D(texTvy, x, y);
	//tv2[pos] = make_float2(x, y);
}

void StereoTgv::FindWarpingVector(const float2 *warpUV, const float *tvx, const float *tvy,
	int w, int h, int s, float2 *tv2)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	// mirror if a coordinate value is out-of-range
	texTvx.addressMode[0] = hipAddressModeMirror;
	texTvx.addressMode[1] = hipAddressModeMirror;
	texTvx.filterMode = hipFilterModeLinear;
	texTvx.normalized = true;

	texTvy.addressMode[0] = hipAddressModeMirror;
	texTvy.addressMode[1] = hipAddressModeMirror;
	texTvy.filterMode = hipFilterModeLinear;
	texTvy.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipBindTexture2D(0, texTvx, tvx, w, h, s * sizeof(float));
	hipBindTexture2D(0, texTvy, tvy, w, h, s * sizeof(float));

	TgvFindWarpingVectorKernel << <blocks, threads >> > (warpUV, w, h, s, tv2);
}


// **************************************************
// ** Find Warping vector direction tv2<float2> for Fisheye Stereo
// **************************************************

__global__ void TgvFindWarpingVectorFloat2Kernel(const float2 *warpUV,
	int width, int height, int stride, float2 *tv2)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	const int pos = ix + iy * stride;

	if (ix >= width || iy >= height) return;

	float x = ((float)ix + warpUV[pos].x + 0.5f) / (float)width;
	float y = ((float)iy + warpUV[pos].y + 0.5f) / (float)height;

	tv2[pos] = tex2D(texTv, x, y);
	//tv2[pos] = make_float2(x, y);
}

void StereoTgv::FindWarpingVector(const float2 *warpUV, const float2 *tv,
	int w, int h, int s, float2 *tv2)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	// mirror if a coordinate value is out-of-range
	texTv.addressMode[0] = hipAddressModeMirror;
	texTv.addressMode[1] = hipAddressModeMirror;
	texTv.filterMode = hipFilterModeLinear;
	texTv.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float2>();

	hipBindTexture2D(NULL, texTv, tv, w, h, s * sizeof(float2));

	TgvFindWarpingVectorFloat2Kernel << <blocks, threads >> > (warpUV, w, h, s, tv2);
}

// **************************************************
// ** Compute Optical flow (u,v) for Fisheye Stereo
// **************************************************

__global__ void TgvComputeOpticalFlowVectorKernel(const float *u, const float2 *tv2,
	int width, int height, int stride, float2 *warpUV)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	const int pos = ix + iy * stride;

	if (ix >= width || iy >= height) return;

	float us = u[pos];
	float2 tv2s = tv2[pos];
	warpUV[pos].x = us * tv2s.x;
	warpUV[pos].y = us * tv2s.y;
}

void StereoTgv::ComputeOpticalFlowVector(const float *u, const float2 *tv2,
	int w, int h, int s, float2 *warpUV)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	TgvComputeOpticalFlowVectorKernel << <blocks, threads >> > (u, tv2, w, h, s, warpUV);
}


// ******************************
// MASKED VeRsIoN
// ******************************
__global__ void TgvWarpingMaskedKernel(float* mask, int width, int height, int stride,
	const float2 *warpUV, float *out)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	if ((iy >= height) && (ix >= width)) return;
	int pos = ix + iy * stride;
	if (mask[pos] == 0.0f) return;

	float x = ((float)ix + warpUV[pos].x + 0.5f) / (float)width;
	float y = ((float)iy + warpUV[pos].y + 0.5f) / (float)height;

	out[pos] = tex2D(texToWarp, x, y);
}

void StereoTgv::WarpImageMasked(const float *src, float* mask, int w, int h, int s,
	const float2 *warpUV, float *out)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	// mirror if a coordinate value is out-of-range
	texToWarp.addressMode[0] = hipAddressModeMirror;
	texToWarp.addressMode[1] = hipAddressModeMirror;
	texToWarp.filterMode = hipFilterModeLinear;
	texToWarp.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipBindTexture2D(0, texToWarp, src, w, h, s * sizeof(float));

	TgvWarpingMaskedKernel << <blocks, threads >> > (mask, w, h, s, warpUV, out);
}

// **************************************************
// ** Find Warping vector direction (tvx2, tvy2) for Fisheye Stereo
// **************************************************

__global__ void TgvFindWarpingVectorMaskedKernel(const float2 *warpUV, float* mask, 
	int width, int height, int stride, float2 *tvx2)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	if ((iy >= height) && (ix >= width)) return;
	int pos = ix + iy * stride;
	if (mask[pos] == 0.0f) return;

	float x = ((float)ix + warpUV[pos].x + 0.5f) / (float)width;
	float y = ((float)iy + warpUV[pos].y + 0.5f) / (float)height;

	tvx2[pos].x = tex2D(texTvx, x, y);
	tvx2[pos].x = tex2D(texTvy, x, y);
	//tv2[pos] = make_float2(x, y);
}

void StereoTgv::FindWarpingVectorMasked(const float2 *warpUV, float* mask, const float *tvx, const float *tvy,
	int w, int h, int s, float2 *tv2)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	// mirror if a coordinate value is out-of-range
	texTvx.addressMode[0] = hipAddressModeMirror;
	texTvx.addressMode[1] = hipAddressModeMirror;
	texTvx.filterMode = hipFilterModeLinear;
	texTvx.normalized = true;

	texTvy.addressMode[0] = hipAddressModeMirror;
	texTvy.addressMode[1] = hipAddressModeMirror;
	texTvy.filterMode = hipFilterModeLinear;
	texTvy.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipBindTexture2D(0, texTvx, tvx, w, h, s * sizeof(float));
	hipBindTexture2D(0, texTvy, tvy, w, h, s * sizeof(float));

	TgvFindWarpingVectorMaskedKernel << <blocks, threads >> > (warpUV, mask, w, h, s, tv2);
}


// **************************************************
// ** Find Warping vector direction tv2<float2> for Fisheye Stereo
// **************************************************

__global__ void TgvFindWarpingVectorFloat2MaskedKernel(const float2 *warpUV, float* mask, 
	int width, int height, int stride, float2 *tv2)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	if ((iy >= height) && (ix >= width)) return;
	int pos = ix + iy * stride;
	if (mask[pos] == 0.0f) return;

	float x = ((float)ix + warpUV[pos].x + 0.5f) / (float)width;
	float y = ((float)iy + warpUV[pos].y + 0.5f) / (float)height;

	tv2[pos] = tex2D(texTv, x, y);
	//tv2[pos] = make_float2(x, y);
}

void StereoTgv::FindWarpingVectorMasked(const float2 *warpUV, float* mask, const float2 *tv,
	int w, int h, int s, float2 *tv2)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	// mirror if a coordinate value is out-of-range
	texTv.addressMode[0] = hipAddressModeMirror;
	texTv.addressMode[1] = hipAddressModeMirror;
	texTv.filterMode = hipFilterModeLinear;
	texTv.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float2>();

	hipBindTexture2D(NULL, texTv, tv, w, h, s * sizeof(float2));

	TgvFindWarpingVectorFloat2MaskedKernel << <blocks, threads >> > (warpUV, mask, w, h, s, tv2);
}

// **************************************************
// ** Compute Optical flow (u,v) for Fisheye Stereo
// **************************************************

__global__ void TgvComputeOpticalFlowVectorMaskedKernel(const float *u, const float2 *tv2, float* mask,
	int width, int height, int stride, float2 *warpUV)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	if ((iy >= height) && (ix >= width)) return;
	int pos = ix + iy * stride;
	if (mask[pos] == 0.0f) return;

	float us = u[pos];
	float2 tv2s = tv2[pos];
	warpUV[pos].x = us * tv2s.x;
	warpUV[pos].y = us * tv2s.y;
}

void StereoTgv::ComputeOpticalFlowVectorMasked(const float *u, const float2 *tv2, float* mask,
	int w, int h, int s, float2 *warpUV)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	TgvComputeOpticalFlowVectorMaskedKernel << <blocks, threads >> > (u, tv2, mask, w, h, s, warpUV);
}